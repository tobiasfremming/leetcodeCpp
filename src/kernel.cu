#include <stdio.h>
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

using namespace std;

// cuda kernel
__global__ void add(int* a, int* b, int* c) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

__managed__ int vector_a[256], vector_b[256], vector_c[256];

int main() {
    cout << "hello world" << endl;
    for (int i = 0; i < 256; i++){
        vector_a[i] = i;
        vector_b[i] = 256 -i;
    }

    add<<<1, 256 >>>(vector_a, vector_b, vector_c);

    hipDeviceSynchronize();

    int result_sum = 0;

    for (int i = 0; i < 256; i++) {
        result_sum += vector_c[i];
    }
    
    printf("Result: sum = % d", result_sum);


    return 0;
}

